#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "sample.cuh"
#include "vector.cuh"
#include "simu_muon.cuh"

extern "C"
{
    float GPU_Sampling_wrapper(double *r,double *pos_x,double *pos_y,double *pos_z, double *intPart, double *fractionPart,double *start_time,double *pmt_x,double *pmt_y,double *pmt_z,double *data_hit,double *data_npe,int *seed,int *size,double* h_result)
    {
        //GPU计时，设置开始和结束事件
        hipEvent_t start, stop;
        hipEvent_t gpu_start,gpu_stop,data_start,data_stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        hipEventCreate(&gpu_start);
        hipEventCreate(&gpu_stop);
        hipEventCreate(&data_start);
        hipEventCreate(&data_stop);
        hipEventRecord(start);
        hipEventRecord(data_start);
        //申请GPU内存
        // double *d_r, *d_pos_x,*d_pos_y,*d_pos_z,*d_intPart,*d_fractionPart,*d_start_time;
        double *d_pmt_x,*d_pmt_y,*d_pmt_z,*d_data_hit,*d_data_npe;
        double *d_result;
        int *d_seed,*d_pmt_res_list;

        CHECK(hipMalloc((double**)&d_pmt_x,size[1]));
        CHECK(hipMalloc((double**)&d_pmt_y,size[1]));
        CHECK(hipMalloc((double**)&d_pmt_z,size[1]));
        CHECK(hipMalloc((double**)&d_data_hit,size[2]));
        CHECK(hipMalloc((double**)&d_data_npe,size[3]));
        CHECK(hipMalloc((int**)&d_seed,size[4]));
        CHECK(hipMalloc((double**)&d_result,pmt_num*pmt_mem*8));
        CHECK(hipMalloc((int**)&d_pmt_res_list,pmt_num*sizeof(int)));

        //设置内存
        CHECK(hipMemset(d_pmt_res_list,0,pmt_num*sizeof(int)));
        CHECK(hipMemset(d_result,0,pmt_num*pmt_mem*8));
        //将CPU内存拷贝到GPU
        CHECK(hipMemcpy(d_pmt_x, pmt_x, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pmt_y, pmt_y, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pmt_z, pmt_z, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_data_hit, data_hit, size[2], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_data_npe, data_npe, size[3], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_seed, seed, size[4], hipMemcpyHostToDevice));
        
        hipEventRecord(data_stop);
        hipEventSynchronize(data_stop);
        // printf("[GPU]GPU数据拷贝完成\n");
        //设置使用编号为0的GPU
        CHECK(hipSetDevice(0));
        // //设置线程数量
        // int threadPerBlock = 1024;
        // int blocksPerGrid =ceil(17746/1024);
        
        // dim3 block(threadPerBlock);
        // //设置块数量
        // dim3 grid(blocksPerGrid);//blocksPerGrid
        int threadPerBlock= 1024;
        int blocksPerGrid = 18;
        dim3 block(threadPerBlock);
        //设置块数量
        dim3 grid(blocksPerGrid);//blocksPerGrid
        // printf("[GPU]网格，线程(%d,%d)\n",blocksPerGrid,threadPerBlock);
        //调用核函数
        hipEventRecord(gpu_start);
        for(int i = 0;i<size[0]/8;i++) {
            CHECK(hipDeviceSynchronize());
            // printf("[GPU]核函数开始运行[%d]\n",i);
            pmt_calculate<<<grid, block>>>(r[i],pos_x[i],pos_y[i],pos_z[i],d_pmt_x,d_pmt_y,d_pmt_z,intPart[i],fractionPart[i],start_time[i],17746,d_data_hit,d_data_npe,(int*)(d_seed+i*pmt_num),d_result,d_pmt_res_list,(int)size[0]/8);
        }
        hipEventRecord(gpu_stop);
        hipEventSynchronize(gpu_stop);
        
        // printf("[GPU]核函数运行完成\n");
        // CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(h_result, d_result, pmt_num*pmt_mem*8, hipMemcpyDeviceToHost));
        
        // printf("threadPerBlock:%d\n",threadPerBlock);
        // printf("blocksPerGrid；%d\n",blocksPerGrid);
        

        //释放GPU内存
        CHECK(hipFree(d_data_hit));
        CHECK(hipFree(d_data_npe));
        CHECK(hipFree(d_pmt_x));
        CHECK(hipFree(d_pmt_y));
        CHECK(hipFree(d_pmt_z));
        CHECK(hipFree(d_seed));
        CHECK(hipFree(d_result));
        // printf("[GPU]GPU运行完成\n");

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float total_time,gputime,datatime;
        //计算用时，精度0.5us
        hipEventElapsedTime(&datatime, data_start, data_stop);
        hipEventElapsedTime(&gputime, gpu_start, gpu_stop);
        hipEventElapsedTime(&total_time, start, stop);
        hipEventDestroy(gpu_start);
        hipEventDestroy(gpu_stop);
        hipEventDestroy(data_start);
        hipEventDestroy(data_stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);        
        printf("total use time %f ms\n", total_time);
        printf("gpu use time %f ms\n",gputime);
        printf("data use time %f ms\n",datatime);
        printf("data transport back use time %f ms\n",total_time - datatime - gputime);
        CHECK(hipDeviceReset());        
        return total_time;
        // return 0.0;
    }
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <string.h>

__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state,int i );

//错误处理宏
#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}

typedef struct
{
	double *d_pmt, *d_hit, *d_result;
	// double *h_result_s;
	hipStream_t stream;
	hipEvent_t start, stop;
}GPU_data;


__global__ void
CDF_Sampling(double *pmt, double *hittime, double *result, int numElements,int max_n,int max_time,int gpu_id)
{
	//compute one-dimensional data index 
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	hiprandState state;
	generateRandomInit(&state,id);
    if (id < numElements)
    {
		double prob; 
		prob = generateRandom(&state);
		double sum = 0;
		int n = 0;
		for (int item = 0; item < max_n; item++)
		{
			sum += pmt[id*max_n+item];
			if (prob <= sum)
			{
				n = item;
				// printf("[gpu: %d] thread %d: hit times:%d\n",gpu_id, id, n);
				break;
			}
		}
		for (int item = 0;item < n;item++) 
		{
			// double prob2;
			prob = generateRandom(&state);
			sum = 0;
			for (int j = 0; j < max_time;j++)
			{
				sum += hittime[id*max_time+j];
				if (prob <= sum)
				{
					result[id*max_n+item] = (double)j;
					// printf("[gpu: %d] thread %d: %dth hit time %d\n",gpu_id, id, item+1,j);
					break;
				}
			}

		}
    }
} 
__device__ double
generateRandom(hiprandState *state)
{
	// int id = blockIdx.x*blockDim.x+threadIdx.x;
    double result = abs(hiprand_uniform_double(state));
	// printf("thread:%d random double: %f \n",id,result);
	return result;
}

__device__ void
generateRandomInit(hiprandState *state,int id)
{
	// long seed = (unsigned long long)clock();
	hiprand_init(id, 0, 0, state);
}
// __host__ void
// initData(double *h_pmt, double *h_hit, int Size)
// {

// }

extern "C" 
{
    float CDF_Sampling_wrapper(double *h_pmt,double *h_hit,double *h_result, int total_num, int nBytes,int max_n,int max_time)
    {
		//GPU计时，设置开始和结束事件
		// cudaEvent_t start, stop;
		// cudaEventCreate(&start);
		// cudaEventCreate(&stop);
		
		
		//获取GPU数量
		int GPU_num;
		CHECK(hipGetDeviceCount(&GPU_num));
		// printf("GPU number:%d\n",GPU_num);
		if(GPU_num<1)
		{
			printf("no CUDA capable devices were detected\n");
			return -1;
		}
		GPU_data data[GPU_num];
		int single_size = total_num/GPU_num;
		int single_bytes = nBytes/GPU_num;
		// cudaEventRecord(start,0);
        //申请GPU内存
		// double *d_pmt, *d_hit,*d_result;
		for(int gpu_id =0; gpu_id < GPU_num; gpu_id++)
		{
			hipSetDevice(gpu_id);
			hipStreamCreate(&data[gpu_id].stream);
			hipEventCreate(&data[gpu_id].start);
			hipEventCreate(&data[gpu_id].stop);
			hipEventRecord(data[gpu_id].start,data[gpu_id].stream);
			CHECK(hipMalloc((double**)&(data[gpu_id].d_pmt),single_bytes));
	    	CHECK(hipMalloc((double**)&(data[gpu_id].d_hit), single_bytes));
			CHECK(hipMalloc((double**)&(data[gpu_id].d_result), single_bytes));
			// data[gpu_id].h_result_s = (double*)malloc(single_bytes);
			//
			CHECK(hipMemcpyAsync(data[gpu_id].d_pmt, (double*)(h_pmt+gpu_id*single_bytes/8), single_bytes, hipMemcpyHostToDevice, data[gpu_id].stream));
			CHECK(hipMemcpyAsync(data[gpu_id].d_hit, (double*)(h_hit+gpu_id*single_bytes/8), single_bytes, hipMemcpyHostToDevice, data[gpu_id].stream));
		}
	    
        //将CPU内存拷贝到GPU
	    // CHECK(cudaMemcpy(d_pmt, h_pmt, nBytes, cudaMemcpyHostToDevice));
		// CHECK(cudaMemcpy(d_hit, h_hit, nBytes, cudaMemcpyHostToDevice));
		
        //设置使用编号为0的GPU
		// CHECK(cudaSetDevice(0));
		
		//设置线程数量
		int threadPerBlock,blocksPerGrid;
		if (single_size<128)
		{
			threadPerBlock = 128;
			blocksPerGrid =1;
		}
		else if(single_size<1024)
		{
			threadPerBlock = 128;
			blocksPerGrid =int(ceil(single_size/(double)threadPerBlock));
		}
		else
		{
			threadPerBlock = 1024;
			blocksPerGrid =int(ceil(single_size/(double)threadPerBlock));
		}
		
	    dim3 block(threadPerBlock);
	    //设置块数量
		dim3 grid(blocksPerGrid);//blocksPerGrid
		
		// cudaEventRecord(gpu_start);
		//调用核函数
		for(int gpu_id = 0; gpu_id < GPU_num; gpu_id++)
		{
			hipSetDevice(gpu_id);
			//第三个参数为0，表示每个block用到的共享内存大小为0
			CDF_Sampling <<<grid, block, 0,data[gpu_id].stream >>>(data[gpu_id].d_pmt, data[gpu_id].d_hit, data[gpu_id].d_result, single_size,max_n,max_time,gpu_id);
			// CHECK(cudaMemcpyAsync(data[gpu_id].h_result_s, data[gpu_id].d_result, single_bytes, cudaMemcpyDeviceToHost,data[gpu_id].stream));
			CHECK(hipMemcpyAsync((double*)(h_result+gpu_id*single_bytes/8),data[gpu_id].d_result,single_bytes,hipMemcpyDeviceToHost,data[gpu_id].stream));
		}
		
		
		
		// cudaEventRecord(gpu_stop);
		// cudaEventSynchronize(gpu_stop);//同步，强制CPU等待GPU event被设定

		CHECK(hipDeviceSynchronize());
		for(int gpu_id = 0; gpu_id < GPU_num; gpu_id++)
		{
			hipEventRecord(data[gpu_id].stop,data[gpu_id].stream);
			hipEventSynchronize(data[gpu_id].stop);
		}
		
		// CHECK(cudaMemcpy(h_result, d_result, nBytes, cudaMemcpyDeviceToHost));
		//等待stream流执行完成
		// for(int gpu_id = 0; gpu_id < GPU_num; gpu_id++)
		// {
		// 	CHECK(cudaStreamSynchronize(data[gpu_id].stream));
		// 	cudaEventRecord(data[gpu_id].stop,data[gpu_id].stream);
		// 	cudaEventSynchronize(data[gpu_id].stop);
		// }
		// for(i  nt gpu_id = 0; gpu_id < GPU_num; gpu_id++)
		// {
		// 	memcpy(h_result+gpu_id*single_bytes,data[gpu_id].h_result_s,single_bytes);
		// }
		// cudaEventRecord(stop,0);
		// cudaEventSynchronize(stop);
		float total_time;
		//计算用时，精度0.5us
		hipEventElapsedTime(&total_time, data[0].start, data[0].stop);
		// cudaEventDestroy(start);
		// cudaEventDestroy(stop);

		// printf("threadPerBlock:%d\n",threadPerBlock);
		// printf("blocksPerGrid；%d\n",blocksPerGrid);
		// printf("total use time %f ms\n", total_time);
		// cudaEventElapsedTime(&time, gpu_start, gpu_stop);
		// cudaEventDestroy(gpu_start);
		// cudaEventDestroy(gpu_stop);
		// printf("gpu use time %f ms\n", time);
		// printf("占用内存：%d B\n", nBytes);
		// printf("占用内存：%d kB\n", nBytes / 1024);

		//释放GPU内存
		for(int gpu_id = 0; gpu_id < GPU_num; gpu_id++)
		{  
			CHECK(hipFree(data[gpu_id].d_pmt));
	    	CHECK(hipFree(data[gpu_id].d_hit));
			CHECK(hipFree(data[gpu_id].d_result));
			CHECK(hipStreamDestroy(data[gpu_id].stream));
			CHECK(hipEventDestroy(data[gpu_id].start));
			CHECK(hipEventDestroy(data[gpu_id].stop));
		}
	  
		CHECK(hipDeviceReset());
		return total_time;
    }
}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state);
//错误处理宏
#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}

__global__ void
CDF_Sampling(double *pmt, double *hittime, double *result, int numElements,int max_n,int max_time)
{
    int id = threadIdx.x;
	hiprandState state;
	generateRandomInit(&state);
    if (id < numElements)
    {
		double prob; 
		prob = generateRandom(&state);
		double sum = 0;
		int n = 0;
		for (int item = 0; item < max_n;item++)
		{
			sum += pmt[id*max_n+item];
			if (prob <= sum)
			{
				n = item;
				printf("thread %d: hit times:%d\n", id, n);
				break;
			}
		}
		for (int item = 0;item < n;item++) 
		{
			double prob2;
			prob2 = generateRandom(&state);
			double sum = 0;
			for (int j = 0; j < max_time;j++)
			{
				sum += hittime[id*max_time+j];
				if (prob2 <= sum)
				{
					result[id*max_n+item] = (double)j;
					printf("thread %d: %dth hit time %d\n", id, item+1,j);
					break;
				}
			}

		}
    }
}
__device__ double
generateRandom(hiprandState *state)
{
	int id = threadIdx.x;
    double result = abs(hiprand_uniform_double(state));
	printf("thread:%d random double: %f \n",id,result);
	return result;
}

__device__ void
generateRandomInit(hiprandState *state)
{
	int id = threadIdx.x;
	long seed = (unsigned long long)clock();
	hiprand_init(seed, id, 0, state);
}

extern "C" 
{
    void CDF_Sampling_wrapper(double *h_pmt,double *h_hit,double *h_result, int total_num, int nBytes,int max_n,int max_time)
    {
		//GPU计时，设置开始和结束事件
		hipEvent_t start, stop, gpu_start,gpu_stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(start);
        //申请GPU内存
	    double *d_pmt, *d_hit,*d_result;
	    CHECK(hipMalloc((double**)&d_pmt,nBytes));
	    CHECK(hipMalloc((double**)&d_hit, nBytes));
	    CHECK(hipMalloc((double**)&d_result, nBytes));
        //将CPU内存拷贝到GPU
	    CHECK(hipMemcpy(d_pmt, h_pmt, nBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_hit, h_hit, nBytes, hipMemcpyHostToDevice));
        //设置使用编号为0的GPU
	    CHECK(hipSetDevice(0));
		//设置线程数量
		int threadPerBlock,blocksPerGrid;
		if (total_num<128)
		{
			threadPerBlock = 128;
			blocksPerGrid =1;
		}
		else if(total_num<1024)
		{
			threadPerBlock = 128;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		else
		{
			threadPerBlock = 1024;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		
	    dim3 block(threadPerBlock);
	    //设置块数量
		dim3 grid(blocksPerGrid);//blocksPerGrid
		
		hipEventRecord(gpu_start);
        //调用核函数
		CDF_Sampling <<<grid, block >>>(d_pmt, d_hit, d_result, total_num,max_n,max_time);
		
		hipEventRecord(gpu_stop);
		hipEventSynchronize(gpu_stop);//同步，强制CPU等待GPU event被设定

        CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(h_result, d_result, nBytes, hipMemcpyDeviceToHost));
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time,total_time;
		//计算用时，精度0.5us
		hipEventElapsedTime(&total_time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);


		printf("total use time %f ms\n", total_time);
		hipEventElapsedTime(&time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		printf("gpu use time %f ms\n", time);
		printf("占用内存：%d B\n", nBytes);
		printf("占用内存：%d kB\n", nBytes / 1024);
        //释放GPU内存
	    CHECK(hipFree(d_pmt));
	    CHECK(hipFree(d_hit));
	    CHECK(hipFree(d_result));
    }
}
#include "hip/hip_runtime.h"
#include "test.cuh"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
__global__ void kernelPrint(){
    printf("GPU run!\n");
}

void CudaRun(){
    printf("cpu run!\n");
    kernelPrint<<<1,5>>>();
    hipDeviceSynchronize();
}

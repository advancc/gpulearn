/**
 * Author:���໴
 * Mail:yiph@ihep.ac.cn
 * Function:Accelerate simulation with GPU
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <string.h>

__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state);

//�������
#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}
//�������
#define CHECK_CURAND(call) \
{\
	const hipError_t error = call;\
	if (error != HIPRAND_STATUS_SUCCESS)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(2);\
	}\
}

//�ں˺��� ��__global__ǰ׺���εĺ����ڱ�������GPU���룬��CPU���ã�����CPUȫ�ֿɼ�
__global__ void
CDF_Sampling(double *pmt, double *hittime, double *result, int numElements)
{
    int id = threadIdx.x;
	hiprandState state;
	generateRandomInit(&state);
    if (id < numElements)
    {
		double prob; 
		prob = generateRandom(&state);
		double sum = 0;
		int n = 0;
		for (int item = 0; item < 10;item++)
		{
			sum += pmt[id*10+item];
			if (prob <= sum)
			{
				n = item;
				printf("thread %d: hit times:%d\n", id, n);
				break;
			}
		}
		for (int item = 0;item < n;item++) 
		{
			double prob2;
			prob2 = generateRandom(&state);
			double sum = 0;
			for (int j = 0; j < 10;j++)
			{
				sum += hittime[id*10+j];
				if (prob2 <= sum)
				{
					result[id*10+item] = (double)j;
					printf("thread %d: %dth hit time %d\n", id, item+1,j);
					break;
				}
			}

		}
    }
}
//GPU���õĺ��� ��__device__ǰ׺���εĺ�����GPU�����У���CPU���ɼ�
__device__ double
generateRandom(hiprandState *state)
{
	int id = threadIdx.x;
    double result = abs(hiprand_uniform_double(state));
	printf("thread:%d random double: %f \n",id,result);
	return result;
}
__device__ void
generateRandomInit(hiprandState *state)
{
	int id = threadIdx.x;
	long seed = (unsigned long long)clock();
	hiprand_init(seed, id, 0, state);
}

/**
 * Host main routine
 */
int
main(void)
{
	
	//���ɼ�����
	int total_num = 10;
	int max_n = 10;
	int max_time = 10;
	size_t nBytes = total_num * max_n * sizeof(double);
	double *pmt;
	pmt = (double*)malloc(nBytes);
	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_n;j++)
		{
			pmt[i*total_num +j] = 0.1;
		}
	}
	double *hittime;
	hittime = (double*)malloc(nBytes);
	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_time;j++)
		{
			hittime[i*total_num+j] = 0.1;
		}
	}
	
	
	double *h_res = (double*)malloc(nBytes);

	//GPU��ʱ�����ÿ�ʼ�ͽ����¼�
	hipEvent_t start, stop, gpu_start,gpu_stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(start);

	//����GPU�ڴ�
	double *d_pmt, *d_hit,*d_result;
	CHECK(hipMalloc((double**)&d_pmt,nBytes));
	CHECK(hipMalloc((double**)&d_hit, nBytes));
	CHECK(hipMalloc((double**)&d_result, nBytes));
	//��CPU�ڴ濽����GPU
	CHECK(hipMemcpy(d_pmt, pmt, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hit, hittime, nBytes, hipMemcpyHostToDevice));

	//����ʹ�ñ��Ϊ0��GPU
	hipSetDevice(0);


	//�����߳�����
	dim3 block(total_num);//threadsPerBlock
	//���ÿ�����
	dim3 grid(total_num / block.x);//blocksPerGrid

	hipEventRecord(gpu_start);

	//���ú˺���
	CDF_Sampling <<<grid, block >>>(d_pmt, d_hit, d_result, total_num);

	hipEventRecord(gpu_stop);
	hipEventSynchronize(gpu_stop);//ͬ����ǿ��CPU�ȴ�GPU event���趨

	
	//��GPU�������ݵ�CPU
	CHECK(hipMemcpy(h_res, d_result, nBytes, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);


	float time;
	//������ʱ������0.5us
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	printf("total use time %f ms\n", time);
	hipEventElapsedTime(&time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	printf("gpu use time %f ms\n", time);

	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_time;j++)
		{
			printf("%f ",h_res[i*10+j]);
		}
		printf("\n");
	}
	//�ͷ�GPU�ڴ�
	CHECK(hipFree(d_pmt));
	CHECK(hipFree(d_hit));
	CHECK(hipFree(d_result));
	free(pmt);
	free(hittime);
	free(h_res);
	//�����ռGPU��Դ
	hipDeviceReset();
	return 0;
 
}


/**
 * Author:易培淮
 * Mail:yiph@ihep.ac.cn
 * Function:Accelerate simulation with GPU
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <string.h>

__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state);

//错误处理宏
#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}
//错误处理宏
#define CHECK_CURAND(call) \
{\
	const hipError_t error = call;\
	if (error != HIPRAND_STATUS_SUCCESS)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(2);\
	}\
}

//内核函数 由__global__前缀修饰的函数在编译生成GPU代码，由CPU调用，并对CPU全局可见
__global__ void
CDF_Sampling(double *pmt, double *hittime, double *result, int numElements)
{
   	int id = threadIdx.x;
	hiprandState state;
	generateRandomInit(&state);
    	if (id < numElements)
    	{
		double prob; 
		prob = generateRandom(&state);
		double sum = 0;
		int n = 0;
		for (int item = 0; item < 10;item++)
		{
			sum += pmt[id*10+item];
			if (prob <= sum)
			{
				n = item;
				printf("thread %d: hit times:%d\n", id, n);
				break;
			}
		}
		for (int item = 0;item < n;item++) 
		{
			double prob2;
			prob2 = generateRandom(&state);
			double sum = 0;
			for (int j = 0; j < 10;j++)
			{
				sum += hittime[id*10+j];
				if (prob2 <= sum)
				{
					result[id*10+item] = (double)j;
					printf("thread %d: %dth hit time %d\n", id, item+1,j);
					break;
				}
			}

		}
    	}
}
//GPU调用的函数 由__device__前缀修饰的函数在GPU上运行，对CPU不可见
__device__ double
generateRandom(hiprandState *state)
{
	int id = threadIdx.x;
    	double result = abs(hiprand_uniform_double(state));
	printf("thread:%d random double: %f \n",id,result);
	return result;
}
__device__ void
generateRandomInit(hiprandState *state)
{
	int id = threadIdx.x;
	long seed = (unsigned long long)clock();
	hiprand_init(seed, id, 0, state);
}

/**
 * Host main routine
 */
int
main(void)
{
	
	//生成假数据
	int total_num = 10;
	int max_n = 10;
	int max_time = 10;
	size_t nBytes = total_num * max_n * sizeof(double);
	double *pmt;
	pmt = (double*)malloc(nBytes);
	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_n;j++)
		{
			pmt[i*max_n +j] = 0.1;
		}
	}
	double *hittime;
	hittime = (double*)malloc(nBytes);
	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_time;j++)
		{
			hittime[i*max_time+j] = 0.1;
		}
	}
	
	
	double *h_res = (double*)malloc(nBytes);

	//GPU计时，设置开始和结束事件
	hipEvent_t start, stop, gpu_start,gpu_stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(start);

	//申请GPU内存
	double *d_pmt, *d_hit,*d_result;
	CHECK(hipMalloc((double**)&d_pmt,nBytes));
	CHECK(hipMalloc((double**)&d_hit, nBytes));
	CHECK(hipMalloc((double**)&d_result, nBytes));
	//将CPU内存拷贝到GPU
	CHECK(hipMemcpy(d_pmt, pmt, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hit, hittime, nBytes, hipMemcpyHostToDevice));

	//设置使用编号为0的GPU
	hipSetDevice(0);


	//设置线程数量
	dim3 block(total_num);//threadsPerBlock
	//设置块数量
	dim3 grid(total_num / block.x);//blocksPerGrid

	hipEventRecord(gpu_start);

	//调用核函数
	CDF_Sampling <<<grid, block >>>(d_pmt, d_hit, d_result, total_num);

	hipEventRecord(gpu_stop);
	hipEventSynchronize(gpu_stop);//同步，强制CPU等待GPU event被设定

	
	//从GPU拷贝数据到CPU
	CHECK(hipMemcpy(h_res, d_result, nBytes, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);


	float time;
	//计算用时，精度0.5us
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	printf("total use time %f ms\n", time);
	hipEventElapsedTime(&time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);
	printf("gpu use time %f ms\n", time);

	for (int i = 0;i < total_num;i++)
	{
		for (int j = 0;j < max_time;j++)
		{
			printf("%f ",h_res[i*max_time+j]);
		}
		printf("\n");
	}
	//释放GPU内存
	CHECK(hipFree(d_pmt));
	CHECK(hipFree(d_hit));
	CHECK(hipFree(d_result));
	free(pmt);
	free(hittime);
	free(h_res);
	//清空所占GPU资源
	hipDeviceReset();
	return 0;
 
}


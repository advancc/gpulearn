/**
* Author:易培淮
* Mail:yiph@ihep.ac.cn
* Function:Accelerate simulation with Single GPU
* 2018/11/27
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
// #include <math.h>
#include <hip/hip_math_constants.h>

// typedef struct arr
// {
//     double *pBase;//存储的是数组第一个元素的地址
//     int len;//数组能容纳的最大元素的个数
//     int cnt;//有效数组个数
//             //自动增长因子
// } Arr;

typedef struct res_arr
{
    double *arr;
    int *pmt_list;
    int index;
    int id;
    // int begin;
    // int len;
} Res_Arr;


__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state,int seed);
__device__ int sampling(hiprandState *state,double *histo,int max,int id);
__device__ int binarySearch(double *histo,double target,int max,int id);
__device__ double calculateAngle(double x,double y,double z,double a,double b,double c);
__device__ void generateHits(double r,double theta, double ratio,double start_time,double *hittime_histo,double *npe,hiprandState *state,Res_Arr r_arr);
__device__ int get_hittime(double r, double theta, int mode, double *hittime_histo, hiprandState *state);
__device__ int get_hittime_bin(int binx, int biny, int mode, double *hittime_histo, hiprandState *state);
__device__ int get_hittime_all(int binx, int biny,double *hittime_histo, hiprandState *state);
__device__ int get_bin_x(double r);
__device__ int get_bin_y(double theta);
__device__ int r_findBin(double r);
__device__ int get_npe(double r,double theta,double *npe,hiprandState *state);
__device__ int r3_findBin(double r3);
__device__ int theta_findBin(double theta);
__device__ int get_npe_num(int binx,int biny,double *npe,hiprandState *state);
__device__ int generateRandomInt(hiprandState *state,int begin,int end);

__global__ void pmt_calculate(double r,double pos_x,double pos_y,double pos_z,double *pmt_x,double *pmt_y,double *pmt_z,double intPart,double fractionPart,double start_time,int numElements,double *hittime_histo,double *npe,int *seed,double *result,int *pmt_res_list,int size);

__device__ void append_res_arr(Res_Arr *p, double val);
__device__ void init_res_arr(Res_Arr *p,double *result,int *pmt_res_list,int pmtid,int size);

// float CDF_Sampling_Wrapping(double *h_pmt,double *h_hit,double *h_result, int *seed,int total_num, int nBytes,int max_n,int max_time);

#define CHECK(call) \
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        printf("Error:%s:%d, ", __FILE__, __LINE__);\
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
}
#define pmt_num 17746
#define pmt_mem 2000
#define HIP_PI_F 3.141592654f



__global__ void
pmt_calculate(double r,double pos_x,double pos_y,double pos_z,double *pmt_x,double *pmt_y,double *pmt_z,double intPart,double fractionPart,double start_time,int numElements,double *hittime_histo,double *npe,int *seed,double *result,int *pmt_res_list,int size){
    int id = blockIdx.x*blockDim.x+threadIdx.x;   
    // printf("num= %d",numElements); 
    // double hittime_single;
    //numElements = pmt numbers
    if (id < numElements){
        hiprandState state;
        generateRandomInit(&state, seed[id]);
        Res_Arr pmt_arr;
        init_res_arr(&pmt_arr,result,pmt_res_list,id,size);
        double theta = calculateAngle(pmt_x[id],pmt_y[id],pmt_z[id],pos_x,pos_y,pos_z);
        // printf("theta = %lf\n",theta);
        for(int j = 0; j < intPart; ++j){
            //r 单位 米
        	generateHits(r,theta,1,start_time,hittime_histo,npe,&state,pmt_arr);
        	// save_hit(&pmt_arr,hittime_single);
        	// save_hits_simple(hittime_single);
        }
        generateHits(r,theta,fractionPart,start_time,hittime_histo,npe,&state,pmt_arr);
        // save_hits_simple(&pmt_arr,hittime_single);
    }
}

__device__ double
calculateAngle(double x,double y,double z,double a,double b,double c)
{
//  printf("x=%lf,y=%lf,z=%lf,a=%lf,b=%lf,c=%lf\n",x,y,z,a,b,c);
    double result = 0;
    if (a == 0 and b == 0 and c == 0){
        return result;
    }
    else{
        result = acos((a*x+b*y+c*z)/(norm3d(x,y,z)*norm3d(a,b,c)));
        //printf("result theta = %lf",result);
        return result;
    }
}

__device__ void 
generateHits(double r,double theta, double ratio,double start_time,double *hittime_histo,double *npe,hiprandState *state,Res_Arr r_arr)
{
    
    int npe_histo_id = get_npe(r,theta,npe,state);
    if (npe_histo_id>0)
    {
        // printf("npe_histo_id = %d,r = %lf,theta = %lf\n",npe_histo_id,r,theta);
        for (int hitj = 0; hitj < npe_histo_id; ++hitj) 
        {
            // printf("ratio=%lf\n",ratio);
            // skip the photon according to the energy deposit
            if (ratio<1 and generateRandom(state)>ratio) 
            {
                continue;
            }
            double hittime_single = start_time;
            // (m_flag_time) 
            hittime_single += (double)get_hittime(r, theta, 0, hittime_histo, state);
            // printf("hittime = %lf\n",hittime_single);
            // generated hit
            // (m_flag_savehits) 
            append_res_arr(&r_arr,hittime_single);
            // save_hits(pmtid, hittime_single,result);
        }
    }
}

 // __device__ void 
 // save_hits(Res_Arr *p,double val){
 // 	append_res_arr(p,val);
 // }

__device__ int
get_hittime(double r, double theta, int mode, double *hittime_histo, hiprandState *state) {
    int binx = get_bin_x(r);
    int biny = get_bin_y(theta);
    
    return get_hittime_bin(binx, biny, mode, hittime_histo, state);
}

__device__ int 
get_hittime_bin(int binx, int biny, int mode, double *hittime_histo, hiprandState *state) {
    // hit time = tmean + tres
    int hittime_single = 0;
    if (mode == 0) {
        hittime_single = get_hittime_all(binx,biny,hittime_histo,state);
    }
    return hittime_single;
}

__device__ int 
get_hittime_all(int binx, int biny,double *hittime_histo, hiprandState *state) {
    // TH1F* h = get_hist(binx, biny);
    const int xbinnum = 200;
    const int ybinnum = 180;
    if (binx<1) { binx = 1; }
    else if (binx > xbinnum) { binx = xbinnum;}
    if (biny<1) { biny = 1; }
    else if (biny > ybinnum) { biny = ybinnum;}
    int idx = (binx-1)*ybinnum+(biny-1);
    int hittime_single = sampling(state,hittime_histo,3000,idx);
    return hittime_single;
}


__device__ int 
get_bin_x(double r) 
{
    int binx = 1;
    int xmode = 2;
    if (xmode == 2) //KR
    {
        binx = r_findBin(r);
    } 
    return binx;
}

__device__ int 
get_bin_y(double theta) {
    int biny = 1;
    int ymode = 4;
    if (ymode == 4) {
        biny = theta_findBin(theta);
    }
    return biny;
}

__device__ int 
r_findBin(double r)
{
    const int binnum = 200;
    const double begin = 0;
    const double end = 17.7;
    if(r==0){
        return 1;
    }
    else{
        return (int)ceil((r-begin)/(end-begin)*binnum); 
    }
}

__device__ int 
get_npe(double r,double theta,double *npe,hiprandState *state)
{
    int binx = r3_findBin(pow(r,3));
    int biny = theta_findBin(theta);
    return get_npe_num(binx,biny,npe,state);
}

__device__ int 
r3_findBin(double r3)
{
    const int binnum = 100;
    const double begin = 0;
    const double end = 5600;
    if(r3 == 0){
        return 1;
    }
    else{
        return (int)ceil((r3-begin)/(end-begin)*binnum);
    }
}


__device__ int 
theta_findBin(double theta)
{
    const int binnum = 180;
    const double begin = 0; 
    const double end = 180.01*HIP_PI_F/180.0;
    if(theta == 0){
        
        return 1;
    }
    else{
        return (int)ceil((theta-begin)/(end-begin)*binnum);
    }
}

__device__ int 
get_npe_num(int binx,int biny,double *npe,hiprandState *state)
{
    int npe_from_single = 0;
    if (1 <= binx and binx <= 100 and 1 <= biny and biny <= 180) {
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));	
    } else if (binx==1 and (biny<1 or biny>180)) {
        biny = generateRandomInt(state,1,180);
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));	
    } else if (binx>1 and (biny<1 or biny>180)) {
        if (biny>180) { biny = 180; }
        else if (biny<1){ biny = 1; }
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));
    } else {
        static long warning = 0;
        ++warning;
        if (warning < 10) {
            printf("npe lost: %d/%d\n", binx,biny);
        } else if (warning == 10) {
            printf("too many npe lost complains.\n");
        }
    }
    return npe_from_single;
}

__device__ double
generateRandom(hiprandState *state)
{
    // int id = blockIdx.x*blockDim.x+threadIdx.x;   
    double result = abs(hiprand_uniform_double(state));
    return result;
}

__device__ int
generateRandomInt(hiprandState *state,int begin,int end)
{
    int result = begin+int(ceil(abs(hiprand_uniform_double(state))*(end-begin)));
    return result;
}

__device__ void
generateRandomInit(hiprandState *state,int seed)
{
    // printf("seed = %d\n",seed);
    // int id = blockIdx.x*blockDim.x+threadIdx.x;   
    hiprand_init(seed, 0, 0, state);
}

__device__ int 
sampling(hiprandState *state,double *histo,int max,int id)
{
    double prob; 
    prob = generateRandom(state);
    return binarySearch(histo,prob,max,id);
}

__device__ int
binarySearch(double *histo,double target,int max,int id)
{

    // int result_for = -1;
    // int result_bin = 0;
    // for (int i=0;i<max;i++){
    //     if (target<=histo[id*max+i]){
    //         // printf("[debug]histo = %lf,%lf\n",histo[id*max],histo[id*max+i]);
    //         // printf("[debug]target=%lf,max=%d,id =%d,i=%d\n",target,max,id,i);
    //         return i;
    //     }
    // }
    // return -1;
 
    int start = 0;
    int end = max-1;
    int mid;
    while(start+1<end){
        mid = start+(end-start)/2;
        if (histo[id*max+mid]==target){
            end = mid;
        } 
        else if (histo[id*max+mid] < target){
            start = mid;
        }
        else if (histo[id*max+mid] > target){
            end = mid;
        }
    }
    if (target <= histo[id*max+start]){
        return start;
    }
    else if (histo[id*max+start] < target){
        return end;
    }
    
    return -1;
}

__device__ void
append_res_arr(Res_Arr *p, double val)//追加，可能成功，可能失败
{
    p->arr[p->index+p->pmt_list[p->id]] = val;
    p->pmt_list[p->id] += 1;
    return;

}

__device__ void
init_res_arr(Res_Arr *p,double *result,int *pmt_res_list,int pmtid,int size){
    p->arr = result;//存储的内存空间
    p->pmt_list = pmt_res_list;//存储每个pmt内存空间使用量
    p->index = pmtid*pmt_mem;//存储该pmt在数组中的起始存取点
    p->id = pmtid;
    // p->begin = begin;
    // p->len = len;
    return;
}

extern "C"
{
    float GPU_Sampling_wrapper(double *r,double *pos_x,double *pos_y,double *pos_z, double *intPart, double *fractionPart,double *start_time,double *pmt_x,double *pmt_y,double *pmt_z,double *data_hit,double *data_npe,int *seed,int *size,double* h_result)
    {
        //GPU计时，设置开始和结束事件
        hipEvent_t start, stop;
        hipEvent_t gpu_start,gpu_stop,data_start,data_stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        hipEventCreate(&gpu_start);
        hipEventCreate(&gpu_stop);
        hipEventCreate(&data_start);
        hipEventCreate(&data_stop);
        hipEventRecord(start);
        hipEventRecord(data_start);
        //申请GPU内存
        // double *d_r, *d_pos_x,*d_pos_y,*d_pos_z,*d_intPart,*d_fractionPart,*d_start_time;
        double *d_pmt_x,*d_pmt_y,*d_pmt_z,*d_data_hit,*d_data_npe;
        double *d_result;
        int *d_seed,*d_pmt_res_list;

        CHECK(hipMalloc((double**)&d_pmt_x,size[1]));
        CHECK(hipMalloc((double**)&d_pmt_y,size[1]));
        CHECK(hipMalloc((double**)&d_pmt_z,size[1]));
        CHECK(hipMalloc((double**)&d_data_hit,size[2]));
        CHECK(hipMalloc((double**)&d_data_npe,size[3]));
        CHECK(hipMalloc((int**)&d_seed,size[4]));
        CHECK(hipMalloc((double**)&d_result,pmt_num*pmt_mem*8));
        CHECK(hipMalloc((int**)&d_pmt_res_list,pmt_num*sizeof(int)));

        //设置内存
        CHECK(hipMemset(d_pmt_res_list,0,pmt_num*sizeof(int)));
        CHECK(hipMemset(d_result,0,pmt_num*pmt_mem*8));
        //将CPU内存拷贝到GPU
        CHECK(hipMemcpy(d_pmt_x, pmt_x, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pmt_y, pmt_y, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_pmt_z, pmt_z, size[1], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_data_hit, data_hit, size[2], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_data_npe, data_npe, size[3], hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_seed, seed, size[4], hipMemcpyHostToDevice));
        
        hipEventRecord(data_stop);
        hipEventSynchronize(data_stop);
        // printf("[GPU]GPU数据拷贝完成\n");
        //设置使用编号为0的GPU
        CHECK(hipSetDevice(0));
        // //设置线程数量
        // int threadPerBlock = 1024;
        // int blocksPerGrid =ceil(17746/1024);
        
        // dim3 block(threadPerBlock);
        // //设置块数量
        // dim3 grid(blocksPerGrid);//blocksPerGrid
        int threadPerBlock= 1024;
        int blocksPerGrid = 18;
        dim3 block(threadPerBlock);
        //设置块数量
        dim3 grid(blocksPerGrid);//blocksPerGrid
        // printf("[GPU]网格，线程(%d,%d)\n",blocksPerGrid,threadPerBlock);
        //调用核函数
        hipEventRecord(gpu_start);
        for(int i = 0;i<size[0]/8;i++) {
            CHECK(hipDeviceSynchronize());
            // printf("[GPU]核函数开始运行[%d]\n",i);
            pmt_calculate<<<grid, block>>>(r[i],pos_x[i],pos_y[i],pos_z[i],d_pmt_x,d_pmt_y,d_pmt_z,intPart[i],fractionPart[i],start_time[i],17746,d_data_hit,d_data_npe,(int*)(d_seed+i*pmt_num),d_result,d_pmt_res_list,(int)size[0]/8);
        }
        hipEventRecord(gpu_stop);
        hipEventSynchronize(gpu_stop);
        
        // printf("[GPU]核函数运行完成\n");
        // CHECK(cudaDeviceSynchronize());

        CHECK(hipMemcpy(h_result, d_result, pmt_num*pmt_mem*8, hipMemcpyDeviceToHost));
        
        // printf("threadPerBlock:%d\n",threadPerBlock);
        // printf("blocksPerGrid；%d\n",blocksPerGrid);
        

        //释放GPU内存
        CHECK(hipFree(d_data_hit));
        CHECK(hipFree(d_data_npe));
        CHECK(hipFree(d_pmt_x));
        CHECK(hipFree(d_pmt_y));
        CHECK(hipFree(d_pmt_z));
        CHECK(hipFree(d_seed));
        CHECK(hipFree(d_result));
        // printf("[GPU]GPU运行完成\n");

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float total_time,gputime,datatime;
        //计算用时，精度0.5us
        hipEventElapsedTime(&datatime, data_start, data_stop);
        hipEventElapsedTime(&gputime, gpu_start, gpu_stop);
        hipEventElapsedTime(&total_time, start, stop);
        hipEventDestroy(gpu_start);
        hipEventDestroy(gpu_stop);
        hipEventDestroy(data_start);
        hipEventDestroy(data_stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);        
        printf("total use time %f ms\n", total_time);
        printf("gpu use time %f ms\n",gputime);
        printf("data use time %f ms\n",datatime);
        printf("data transport back use time %f ms\n",total_time - datatime - gputime);
        CHECK(hipDeviceReset());        
        return total_time;
        // return 0.0;
    }
}
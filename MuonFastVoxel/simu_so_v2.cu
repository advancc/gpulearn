#include "hip/hip_runtime.h"
/**
 * Author:易培淮
 * Mail:yiph@ihep.ac.cn
 * Function:Accelerate simulation with Single GPU
 * 2018/11/27
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state,int seed);
__device__ int sampling(hiprandState state,double *histo,int max,int id);
float CDF_Sampling_Wrapping(double *h_pmt,double *h_hit,double *h_result, int *seed,int total_num, int nBytes,int max_n,int max_time);

#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}

__global__ void
CDF_Sampling(double *pmt, double *hittime, double *result, int *seed,int numElements,int max_n,int max_time)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState state;
    generateRandomInit(&state,seed[id]);
    if (id < numElements)
    {
        int n = sampling(state,pmt,max_n,id);
		for (int item = 0;item < n;item++) 
		{
			result[id*max_n+item] = (double)sampling(state,hittime,max_time,id);
		}
    }
}
__device__ double
generateRandom(hiprandState *state)
{
    double result = abs(hiprand_uniform_double(state));
    return result;
}
__device__ void
generateRandomInit(hiprandState *state,int seed)
{
    hiprand_init(seed, 0, 0, state);
}
__device__ int 
sampling(hiprandState state,double *histo,int max,int id)
{
    double prob; 
	prob = generateRandom(&state);
	return binarySearch(histo,prob,max,id);
	// int result = 0;
	// return result;
	// double sum = 0;
    // for (int item = 0; item < max;item++)
    // {
    //     sum += histo[id*max+item];
    //     if (prob <= sum)
    //     {
    //         result = item;
    //         printf("thread %d: hit times:%d\n", id, result);
    //         break;
    //     }
    // }
    // return result;
}
__device__ int
binarySearch(double *histo,double target,int max,int id)
{
	int start = 0;
	int end = max-1; 
	int mid;

	while(start+1<end){
		mid = start+(end-start)/2;
		if (histo[id*max+mid]==target){
			end = mid;
		} 
		else if (histo[id*max+mid] < target){
			start = mid;
		}
		else if (histo[id*max+mid] > target){
			end = mid;
		}
	}
	if (histo[id*max+start] == target){
		return start;
	}
	if (histo[id*max+end] == target){
		return end;
	}
	return -1
}

extern "C" 
{
    float CDF_Sampling_wrapper(double *h_pmt,double *h_hit,double *h_result, int *seed, int total_num, int nBytes,int max_n,int max_time)
    {
		//GPU计时，设置开始和结束事件
		hipEvent_t start, stop, gpu_start,gpu_stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(start);
        //申请GPU内存
		double *d_pmt, *d_hit,*d_result;
		int *d_seed;
	    CHECK(hipMalloc((double**)&d_pmt,nBytes));
	    CHECK(hipMalloc((double**)&d_hit, nBytes));
		CHECK(hipMalloc((double**)&d_result, nBytes));
		CHECK(hipMalloc((int**)&d_seed,nBytes/2));
        //将CPU内存拷贝到GPU
	    CHECK(hipMemcpy(d_pmt, h_pmt, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_hit, h_hit, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_seed,seed,nBytes/2,hipMemcpyHostToDevice));
        //设置使用编号为0的GPU
	    CHECK(hipSetDevice(0));
		//设置线程数量
		int threadPerBlock,blocksPerGrid;
		if (total_num<128)
		{
			threadPerBlock = 128;
			blocksPerGrid =1;
		}
		else if(total_num<1024)
		{
			threadPerBlock = 128;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		else
		{
			threadPerBlock = 1024;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		
	    dim3 block(threadPerBlock);
	    //设置块数量
		dim3 grid(blocksPerGrid);//blocksPerGrid
		
		hipEventRecord(gpu_start);
        //调用核函数
		CDF_Sampling <<<grid, block >>>(d_pmt, d_hit, d_result, d_seed,total_num,max_n,max_time);
		
		hipEventRecord(gpu_stop);
		hipEventSynchronize(gpu_stop);//同步，强制CPU等待GPU event被设定

        CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(h_result, d_result, nBytes, hipMemcpyDeviceToHost));
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time,total_time;
		//计算用时，精度0.5us
		hipEventElapsedTime(&total_time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		printf("threadPerBlock:%d\n",threadPerBlock);
		printf("blocksPerGrid；%d\n",blocksPerGrid);
		printf("total use time %f ms\n", total_time);
		hipEventElapsedTime(&time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		printf("gpu use time %f ms\n", time);
		printf("占用内存：%d B\n", nBytes);
		printf("占用内存：%d kB\n", nBytes / 1024);
        //释放GPU内存
	    CHECK(hipFree(d_pmt));
	    CHECK(hipFree(d_hit));
		CHECK(hipFree(d_result));
		return total_time;
    }
}


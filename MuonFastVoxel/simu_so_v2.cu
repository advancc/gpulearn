#include "hip/hip_runtime.h"
/**
 * Author:易培淮
 * Mail:yiph@ihep.ac.cn
 * Function:Accelerate simulation with GPU
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>


__device__ double generateRandom(hiprandState *state);
__device__ void generateRandomInit(hiprandState *state,int id);
__device__ int sampling(hiprandState state,double *histo,int max,int id);
float CDF_Sampling_Wrapping(double *h_pmt,double *h_hit,double *h_result, int total_num, int nBytes,int max_n,int max_time);

#define CHECK(call) \
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		printf("Error:%s:%d, ", __FILE__, __LINE__);\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
		exit(1);\
	}\
}

__global__ void
CDF_Sampling(hiprandState *global_state,double *pmt, double *hittime, double *result, int numElements,int max_n,int max_time)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState state;
    generateRandomInit(&state,id);
    if (id < numElements)
    {
        int n = sampling(state,pmt,max_n,id);
		for (int item = 0;item < n;item++) 
		{
			result[id*max_n+item] = (double)sampling(state,hittime,max_time,id);
		}
    }
}
__device__ double
generateRandom(hiprandState *state)
{
    double result = abs(hiprand_uniform_double(state));
    return result;
}
__device__ void
generateRandomInit(hiprandState *state,int id)
{
    hiprand_init(id, 0, 0, state);
}
__device__ int 
sampling(hiprandState state,double *histo,int max,int id)
{
    double prob; 
    prob = generateRandom(&state);
    double sum = 0;
    int result = 0;
    for (int item = 0; item < max;item++)
    {
        sum += histo[id*max+item];
        if (prob <= sum)
        {
            result = item;
            // printf("thread %d: hit times:%d\n", id, n);
            break;
        }
    }
    return result;
}
extern "C" 
{
    float CDF_Sampling_wrapper(double *h_pmt,double *h_hit,double *h_result, int total_num, int nBytes,int max_n,int max_time)
    {
		//GPU计时，设置开始和结束事件
		hipEvent_t start, stop, gpu_start,gpu_stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventCreate(&gpu_start);
		hipEventCreate(&gpu_stop);
		hipEventRecord(start);
        //申请GPU内存
	    double *d_pmt, *d_hit,*d_result;
	    CHECK(hipMalloc((double**)&d_pmt,nBytes));
	    CHECK(hipMalloc((double**)&d_hit, nBytes));
	    CHECK(hipMalloc((double**)&d_result, nBytes));
        //将CPU内存拷贝到GPU
	    CHECK(hipMemcpy(d_pmt, h_pmt, nBytes, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_hit, h_hit, nBytes, hipMemcpyHostToDevice));
        //设置使用编号为0的GPU
	    CHECK(hipSetDevice(0));
		//设置线程数量
		int threadPerBlock,blocksPerGrid;
		if (total_num<128)
		{
			threadPerBlock = 128;
			blocksPerGrid =1;
		}
		else if(total_num<1024)
		{
			threadPerBlock = 128;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		else
		{
			threadPerBlock = 1024;
			blocksPerGrid =int(ceil(total_num/(double)threadPerBlock));
		}
		
	    dim3 block(threadPerBlock);
	    //设置块数量
		dim3 grid(blocksPerGrid);//blocksPerGrid
		
		hipEventRecord(gpu_start);
        //调用核函数
		CDF_Sampling <<<grid, block >>>(d_pmt, d_hit, d_result, total_num,max_n,max_time);
		
		hipEventRecord(gpu_stop);
		hipEventSynchronize(gpu_stop);//同步，强制CPU等待GPU event被设定

        CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(h_result, d_result, nBytes, hipMemcpyDeviceToHost));
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time,total_time;
		//计算用时，精度0.5us
		hipEventElapsedTime(&total_time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		printf("threadPerBlock:%d\n",threadPerBlock);
		printf("blocksPerGrid；%d\n",blocksPerGrid);
		printf("total use time %f ms\n", total_time);
		hipEventElapsedTime(&time, gpu_start, gpu_stop);
		hipEventDestroy(gpu_start);
		hipEventDestroy(gpu_stop);
		printf("gpu use time %f ms\n", time);
		printf("占用内存：%d B\n", nBytes);
		printf("占用内存：%d kB\n", nBytes / 1024);
        //释放GPU内存
	    CHECK(hipFree(d_pmt));
	    CHECK(hipFree(d_hit));
		CHECK(hipFree(d_result));
		return total_time;
    }
}

